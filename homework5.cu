/*
Bryce Taylor
bktaylor2@crimson.ua.edu
CS 581
Homework #5

To compile:
nvcc homework5.cu -o homework5

To run:
./homework5 (Size of board) (Max generations) (Output file directory)
./homework5 5000 5000 outputs
*/


#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>

using namespace std;

// Main Game of Life kernel
__global__ void mainKernel(int* curBoard, int* newBoard, int realSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure cell is not a ghost cell
    if (row > 0 && row < realSize - 1 && col > 0 && col < realSize - 1) {
        // Logic to find number of neighbors and calculate new cell
        int numNeighbors = 0;
        numNeighbors += curBoard[(row-1)*realSize + col-1] + curBoard[(row-1)*realSize + col] + curBoard[(row-1)*realSize + col+1] + \
                        curBoard[(row+1)*realSize + col-1] + curBoard[(row+1)*realSize + col] + curBoard[(row+1)*realSize + col+1] + \
                        curBoard[(row)*realSize + col-1] + curBoard[(row)*realSize + col+1];
        if (curBoard[row * realSize + col] == 0) {
            if (numNeighbors == 3) {
                newBoard[row * realSize + col] = 1;
            }
            else {
                newBoard[row * realSize + col] = 0;
            }
        }
        else if (curBoard[row * realSize + col] == 1) {
            if (numNeighbors < 2) {
                newBoard[row * realSize + col] = 0;
            }
            else if (numNeighbors > 3) {
                newBoard[row * realSize + col] = 0;
            }
            else {
                newBoard[row * realSize + col] = 1;
            }
        }
    }
}

int main(int argc, char* argv[]) {
    int boardSize; // Board size (N)
    int realSize;
    int maxGenerations; // Max # of iterations
    string outputDir; // Directory to write output file to

    srand(0); // Set seed
    
    // Input processing
    boardSize = atoi(argv[1]);
    realSize = atoi(argv[1]) + 2;
    maxGenerations = atoi(argv[2]);
    outputDir = argv[3];

    // Allocate and initialize board
    int* curBoard = (int*)malloc(realSize * realSize * sizeof(int));
    for (int i = 0; i < realSize * realSize; i++) {
        curBoard[i] = 0;
    }
    for (int i = 1; i < realSize - 1; i++) {
        for (int j = 1; j < realSize - 1; j++) {
            curBoard[(i * realSize) + j] = rand() % 2;
        }
    }

    // Allocate and copy device memory
    int* devCur;
    int* devNew;
    hipMalloc(&devCur, realSize * realSize * sizeof(int));
    hipMalloc(&devNew, realSize * realSize * sizeof(int));
    hipMemcpy(devCur, curBoard, realSize * realSize * sizeof(int), hipMemcpyHostToDevice);

    // Initialize block and thread size
    int blockSize = 16;
    dim3 threads(blockSize, blockSize);
    dim3 blocks((boardSize + blockSize - 1) / blockSize, (boardSize + blockSize - 1) / blockSize);

    // Start timer
    auto start = chrono::high_resolution_clock::now();

    // Main algorithm loop
    for (int n = 1; n <= maxGenerations; n++) {
        // Call main kernel for game of life logic
        mainKernel<<<blocks, threads>>>(devCur, devNew, realSize);
        hipDeviceSynchronize();
        hipMemcpy(devCur, devNew, realSize * realSize * sizeof(int), hipMemcpyDeviceToDevice);
    }
    // Copy board back to host
    hipMemcpy(curBoard, devCur, realSize * realSize * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // End timer and calculate time taken
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end - start);

    cout << "Time taken: " << duration.count() << " ms" << endl;
    
    // Write output to a file
    string outputFileName = outputDir + "/gpu_" + to_string(boardSize) + "_" + to_string(maxGenerations) + ".txt";
    ofstream OutputFile(outputFileName);
    for (int i = 0; i < realSize; i++) {
        for (int j = 0; j < realSize; j++) {
            OutputFile << curBoard[(i * realSize) + j] << " ";
        }
        OutputFile << endl;
    }
    OutputFile.close();

    return 0;
}
